#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// Copyright 2004-present Facebook. All Rights Reserved.
#include <hip/hip_runtime.h>
#include <cassert>
#include "fbgemm_gpu/cuda_utils.cuh"
#include "fbgemm_gpu/quantize_ops.cuh"
#include "fbgemm_gpu/quantize_wrappers.cuh"

void fbgemm_gpu_test::FloatToFused8BitRowwiseQuantized(
    const int32_t nrows,
    const int32_t ncols,
    const float* __restrict__ input,
    uint8_t* __restrict__ output) {
  int threads_per_block = 256;
  int num_blocks = (nrows + threads_per_block - 1) / threads_per_block;

  if (nrows <= 20) {
    _float_to_fused8bitrowwise_cuda_kernel<<<num_blocks, threads_per_block>>>(
        input, nrows, ncols, output);
  } else {
    float* range_tensor;
    CUDA_CHECK(hipMalloc((void**)&range_tensor, (nrows) * sizeof(float)));
    _get_8bit_qparam_cuda_kernel<<<num_blocks, threads_per_block>>>(
        input, nrows, ncols, output, range_tensor);

    int blockDim_x = std::min(ncols, threads_per_block);
    dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);
    int gridDim_x = (ncols + blockDim.x - 1) / blockDim.x;
    int gridDim_y = std::min((nrows + blockDim.y - 1) / blockDim.y, 65535u);
    dim3 gridDim(gridDim_x, gridDim_y);

    _compute_8bit_quantize_cuda_kernel<<<gridDim, blockDim>>>(
        input, range_tensor, nrows, ncols, output);
    CUDA_CHECK(hipFree(range_tensor));
  }
  CUDA_CHECK(hipGetLastError());
}

void fbgemm_gpu_test::Fused8BitRowwiseQuantizedToFloat(
    const int32_t nrows,
    const int32_t ncols,
    const uint8_t* __restrict__ input,
    float* __restrict__ output) {
  int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  int output_columns = ncols_aligned - 2 * sizeof(float);

  int threads_per_block = 256;

  int blockDim_x = std::min(threads_per_block, output_columns);
  dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);

  int gridDim_x = (output_columns + blockDim.x - 1) / blockDim.x;
  int gridDim_y = std::min((nrows + blockDim.y - 1) / blockDim.y, 65535u);
  dim3 gridDim(gridDim_x, gridDim_y);

  _fused8bitrowwise_to_float_cuda_kernel<<<gridDim, blockDim>>>(
      input, nrows, ncols, output);

  CUDA_CHECK(hipGetLastError());
}

void fbgemm_gpu_test::FloatToFusedNBitRowwiseQuantizedSBHalf(
    const int32_t nrows,
    const int32_t ncols,
    const int32_t bit_rate,
    const float* __restrict__ input,
    uint8_t* __restrict__ output) {
  assert(
      ncols % (2 * (8 / bit_rate)) == 0 &&
      "ncols needs to be multiple of 2 Bytes (half type size) to make the address aligned");

  int threads_per_block = 256;
  int num_blocks = (nrows + threads_per_block - 1) / threads_per_block;
  // think unsigned as we use 0, 255

  _float_to_fusednbitrowwise_cuda_kernel<<<num_blocks, threads_per_block>>>(
      bit_rate, input, nrows, ncols, output);

  CUDA_CHECK(hipGetLastError());
}

void fbgemm_gpu_test::FusedNBitRowwiseQuantizedSBHalfToFloat(
    const int32_t nrows,
    const int32_t ncols,
    const int32_t bit_rate,
    const uint8_t* __restrict__ input,
    float* __restrict__ output) {
  int num_elem_per_byte = 8 / bit_rate;
  int output_columns = (ncols - 2 * sizeof(__half)) * num_elem_per_byte;

  int threads_per_block = 256;

  int blockDim_x = std::min(output_columns, threads_per_block);
  dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);
  int gridDim_x = (output_columns + blockDim.x - 1) / blockDim.x;
  int gridDim_y = std::min((nrows + blockDim.y - 1) / blockDim.y, 65535u);
  dim3 gridDim(gridDim_x, gridDim_y);

  _fusednbitrowwise_to_float_cuda_kernel<<<gridDim, blockDim>>>(
      bit_rate, input, nrows, ncols, output);

  CUDA_CHECK(hipGetLastError());
}

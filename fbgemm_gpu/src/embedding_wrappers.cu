#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// Copyright 2004-present Facebook. All Rights Reserved.
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include "fbgemm_gpu/batched_unary_embeddings.cuh"
#include "fbgemm_gpu/cuda_utils.cuh"
#include "fbgemm_gpu/embedding_wrappers.cuh"

void fbgemm_gpu_test::batched_unary_embeddings_forward(
    const int32_t N,
    const int32_t B,
    const int32_t T,
    const float* __restrict__ weight,
    const long* __restrict__ table_offsets,
    const long* __restrict__ offsets,
    const long* __restrict__ indices,
    float* __restrict__ output) {
  int32_t threads = std::min<int32_t>(B, 512);
  dim3 blocks((B + threads - 1) / threads, T, N);
  assert(T <= 65535);
  assert(N <= 65535);
  batched_unary_embeddings_forward_kernel<float><<<blocks, threads>>>(
      N, B, T, weight, table_offsets, offsets, indices, output);
  CUDA_CHECK(hipGetLastError());
}

void fbgemm_gpu_test::batched_unary_embeddings_backward(
    const int32_t N,
    const int32_t B,
    const int32_t T,
    const float* __restrict__ grad_output,
    const long* __restrict__ table_offsets,
    const long* __restrict__ offsets,
    const long* __restrict__ indices,
    float* __restrict__ grad_weight) {
  int threads = std::min<int32_t>(N * T, 512);
  dim3 blocks((N * T + threads - 1) / threads);
  batched_unary_embeddings_backward_kernel<float><<<blocks, threads>>>(
      N, B, T, grad_output, table_offsets, offsets, indices, grad_weight);
  CUDA_CHECK(hipGetLastError());
}
